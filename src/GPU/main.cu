#include "hip/hip_runtime.h"
#include "camera.h"
#include "sphere.h"
#include "texture.h"
#include "quad.h"
#include "constant_medium.h"
// #include "bvh.h"

#include <chrono>

#define MAX_OBJS 4000

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '"
                  << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_world1(hittable **d_list, hittable_list **d_world, camera **cam, int image_width,
                             int image_height, hiprandState *devStates, int samples_per_pixel, int max_depth, bool use_bvh)
{
    hiprandState *local_rand_state = &devStates[0];

    auto checker = new checker_texture(0.32, color(.8, .1, .1), color(.9, .9, .9));

    d_list[0] = new sphere(point3(0.0f, -1000.0f, -1.0f), 1000.0f, new lambertian(checker));
    int i = 1;
    for (int a = -11; a < 11; a++)
    {
        for (int b = -11; b < 11; b++)
        {
            float choose_mat = random_float(local_rand_state);
            point3 center(a + random_float(local_rand_state), 0.2f, b + random_float(local_rand_state));
            if (choose_mat < 0.8f)
            {
                d_list[i++] =
                    new sphere(center, 0.2f,
                               new lambertian(color(random_float(local_rand_state) * random_float(local_rand_state),
                                                    random_float(local_rand_state) * random_float(local_rand_state),
                                                    random_float(local_rand_state) * random_float(local_rand_state))));
            }
            else if (choose_mat < 0.95f)
            {
                d_list[i++] = new sphere(center, 0.2f,
                                         new metal(color(0.5f * (1.0f + random_float(local_rand_state)),
                                                         0.5f * (1.0f + random_float(local_rand_state)),
                                                         0.5f * (1.0f + random_float(local_rand_state))),
                                                   0.5f * random_float(local_rand_state)));
            }
            else
            {
                d_list[i++] = new sphere(center, 0.2f, new dielectric(1.5));
            }
        }
    }
    d_list[i++] = new sphere(point3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));
    d_list[i++] = new sphere(point3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(color(0.4f, 0.2f, 0.1f)));
    d_list[i++] = new sphere(point3(4.0f, 1.0f, 0.0f), 1.0f, new metal(color(0.7f, 0.6f, 0.5f), 0.0f));
    
    *d_world = new hittable_list(d_list, i);

    *cam = new camera(image_width, image_height, samples_per_pixel, max_depth, 20.0f, point3(13.0f, 2.0f, 3.0f), point3(0.0f, 0.0f, 0.0f),
                      vec3(0.0f, 1.0f, 0.0f), 0.6f, 10.0f, color(0.70, 0.80, 1.00));
}

__global__ void create_world2(hittable **d_list, hittable_list **d_world, camera **cam, int image_width,
                             int image_height, hiprandState *devStates, int samples_per_pixel, int max_depth, bool use_bvh)
{
    // Cornell box sides
    int i = 0;
    d_list[i++] = new quad(point3(555,0,0), vec3(0,0,555), vec3(0,555,0), new lambertian(color(.12, .45, .15)));
    d_list[i++] = new quad(point3(0,0,555), vec3(0,0,-555), vec3(0,555,0), new lambertian(color(.65, .05, .05)));
    d_list[i++] = new quad(point3(0,555,0), vec3(555,0,0), vec3(0,0,555), new lambertian(color(.73, .73, .73)));
    d_list[i++] = new quad(point3(0,0,555), vec3(555,0,0), vec3(0,0,-555), new lambertian(color(.73, .73, .73)));
    d_list[i++] = new quad(point3(555,0,555), vec3(-555,0,0), vec3(0,555,0), new lambertian(color(.73, .73, .73)));

    // Light
    d_list[i++] = new quad(point3(213,554,227), vec3(130,0,0), vec3(0,0,105), new diffuse_light(color(15, 15, 15)));

    // Box
    d_list[i++] = new quad(point3(265, 0, 295), vec3(159.38, 0, -42.71), vec3(42.71, 0, 159.38), new metal(color(0.8, 0.85, 0.88), 0.0));
    d_list[i++] = new quad(point3(424.38, 0, 252.29), vec3(42.71, 0, 159.38), vec3(0, 330, 0), new metal(color(0.8, 0.85, 0.88), 0.0));
    d_list[i++] = new quad(point3(467.08, 0, 411.67), vec3(-159.38, 0, 42.71), vec3(0, 330, 0), new metal(color(0.8, 0.85, 0.88), 0.0)); 
    d_list[i++] = new quad(point3(307.71, 0, 454.38), vec3(-42.71, 0, -159.38), vec3(0, 330, 0), new metal(color(0.8, 0.85, 0.88), 0.0));
    d_list[i++] = new quad(point3(265, 330, 295), vec3(159.38, 0, -42.71), vec3(42.71, 0, 159.38), new metal(color(0.8, 0.85, 0.88), 0.0)); 
    d_list[i++] = new quad(point3(265, 0, 295), vec3(159.38, 0, -42.71), vec3(0, 330, 0), new metal(color(0.8, 0.85, 0.88), 0.0));   


    // Glass Sphere
    d_list[i++] = new sphere(point3(190.0f,90.0f,190.0f), 90.0f, new dielectric(1.5f));
    
    *d_world = new hittable_list(d_list, i);

    *cam = new camera(image_width, image_height, samples_per_pixel, max_depth, 40.0f, point3(278.0f, 278.0f, -800.0f), point3(278.0f, 278.0f, 0.0f),
                      vec3(0.0f, 1.0f, 0.0f), 0.0f, 10.0f, color(0,0,0));
}

__global__ void create_world3(hittable **d_list, hittable_list **d_world, camera **cam, int image_width,
                             int image_height, hiprandState *devStates, int samples_per_pixel, int max_depth, bool use_bvh)
{
    hiprandState *local_rand_state = &devStates[0];

    int i = 0;
    int boxes_per_side = 20;
    for (int k = 0; k < boxes_per_side; k++) {
        for (int j = 0; j < boxes_per_side; j++) {
            auto w = 100.0;
            auto x0 = -1000.0 + k*w;
            auto z0 = -1000.0 + j*w;
            auto y0 = 0.0;
            auto x1 = x0 + w;
            auto y1 = random_int(1, 101, local_rand_state);
            auto z1 = z0 + w;

            point3 a = point3(x0,y0,z0);
            point3 b = point3(x1,y1,z1);
            auto min = point3(fminf(a.x(),b.x()), fminf(a.y(),b.y()), fminf(a.z(),b.z()));
            auto max = point3(fmaxf(a.x(),b.x()), fmaxf(a.y(),b.y()), fmaxf(a.z(),b.z()));

            auto dx = vec3(max.x() - min.x(), 0, 0);
            auto dy = vec3(0, max.y() - min.y(), 0);
            auto dz = vec3(0, 0, max.z() - min.z());

            d_list[i++] = new quad(point3(min.x(), min.y(), max.z()),  dx,  dy, new lambertian(color(0.48, 0.83, 0.53))); // front
            d_list[i++] = new quad(point3(max.x(), min.y(), max.z()), -dz,  dy, new lambertian(color(0.48, 0.83, 0.53))); // right
            d_list[i++] = new quad(point3(max.x(), min.y(), min.z()), -dx,  dy, new lambertian(color(0.48, 0.83, 0.53))); // back
            d_list[i++] = new quad(point3(min.x(), min.y(), min.z()),  dz,  dy, new lambertian(color(0.48, 0.83, 0.53))); // left
            d_list[i++] = new quad(point3(min.x(), max.y(), max.z()),  dx, -dz, new lambertian(color(0.48, 0.83, 0.53))); // top
            d_list[i++] = new quad(point3(min.x(), min.y(), min.z()),  dx,  dz, new lambertian(color(0.48, 0.83, 0.53))); // bottom
        }
    }

    // Light
    d_list[i++] = new quad(point3(123,554,147), vec3(300,0,0), vec3(0,0,265), new diffuse_light(color(7, 7, 7)));

    d_list[i++] = new sphere(point3(400, 400, 200), 50, new lambertian(color(0.7, 0.3, 0.1)));
    d_list[i++] = new sphere(point3(260, 150, 45), 50, new dielectric(1.5));
    d_list[i++] = new sphere(point3(0, 150, 145), 50, new metal(color(0.8, 0.8, 0.9), 1.0));

    auto boundary = new sphere(point3(360,150,145), 70, new dielectric(1.5));
    d_list[i++] = boundary;
    d_list[i++] = new constant_medium(boundary, 0.2, color(0.2, 0.4, 0.9));

    boundary = new sphere(point3(0,0,0), 5000, new dielectric(1.5));
    d_list[i++] = new constant_medium(boundary, .0001, color(1,1,1));

    auto checker = new checker_texture(0.32, color(.8, .1, .1), color(.9, .9, .9));
    d_list[i++] = new sphere(point3(400,200,400), 100, new lambertian(checker));

    auto pertext = new noise_texture(0.2, local_rand_state);
    d_list[i++] = new sphere(point3(220,280,300), 80, new lambertian(pertext));

    int ns = 1000;
    for (int j = 0; j < ns; j++) {
        auto position = point3::random(local_rand_state,0,165) + point3(-100, 270, 395);
        point3 position1 = point3(position.x(), position.y(), position.z());
        d_list[i++] = new sphere(position1, 10, new lambertian(color(.73, .73, .73)));
    }

    *d_world = new hittable_list(d_list, i);

    *cam = new camera(image_width, image_height, samples_per_pixel, max_depth, 40.0f, point3(478, 278, -600), point3(278, 278, 0),
                      vec3(0.0f, 1.0f, 0.0f), 0.0f, 10.0f, color(0,0,0));
}


__global__ void free_world(hittable **d_list, hittable_list **d_world, camera **d_camera)
{
    for (int i = 0; i < (*d_world)->obj_num; i++)
    {
        if (d_list[i]->get_type() == HittableType::SPHERE) {
            delete ((sphere*)d_list[i])->get_mat();     
        } else if (d_list[i]->get_type() == HittableType::QUAD) {
            delete ((quad*)d_list[i])->get_mat();          
        } else if (d_list[i]->get_type() == HittableType::MEDIUM) {
            delete ((constant_medium*)d_list[i])->get_mat();          
        }
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void call_render(hittable_list **d_world, camera **cam, int image_width, int image_height, uint8_t *output,
                            hiprandState *devStates)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= image_width) || (j >= image_height))
        return;

    hiprandState *local_rand_state = &devStates[j * image_width + i];

    (*cam)->render(d_world, i, j, local_rand_state, output);
}

__global__ void rand_init(hiprandState *rand_state, unsigned long seed) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(seed, 0, 0, rand_state);
    }
}

int main()
{
    int image_width = 1080;
    int image_height = 720;
    int samples_per_pixel = 10000;
    int max_depth = 50;
    int scene = 3;
    bool use_bvh = false; // TODO: Fix the dynamic memory allocation problems

    int total_pixels = image_width * image_height;

    hiprandState *devStates;
    checkCudaErrors(hipMalloc((void **)&devStates, total_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, 1*sizeof(hiprandState)));

    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, MAX_OBJS * sizeof(hittable *)));
    hittable_list **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable_list *)));
    uint8_t *d_output;
    checkCudaErrors(hipMalloc((void **)&d_output, total_pixels * 3 * sizeof(uint8_t)));
    camera **cam;
    checkCudaErrors(hipMalloc((void **)&cam, sizeof(camera)));

    int blockdimx = 16;
    int blockdimy = 16;
    dim3 gridSize((image_width + blockdimx - 1) / blockdimx, (image_height + blockdimy - 1) / blockdimy);
    dim3 blockSize(blockdimx, blockdimy);

    auto start_time = std::chrono::high_resolution_clock::now();

    unsigned long seed = 1984;
    rand_init<<<1, 1>>>(d_rand_state, seed);
    init_random_state<<<gridSize, blockSize>>>(devStates, image_width, image_height, seed);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    switch (scene) {
        case 1: create_world1<<<1, 1>>>(d_list, d_world, cam, image_width, image_height, d_rand_state, samples_per_pixel, max_depth, use_bvh); break;
        case 2: create_world2<<<1, 1>>>(d_list, d_world, cam, image_width, image_height, d_rand_state, samples_per_pixel, max_depth, use_bvh); break;
        case 3: create_world3<<<1, 1>>>(d_list, d_world, cam, image_width, image_height, d_rand_state, samples_per_pixel, max_depth, use_bvh); break;
    }
    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    call_render<<<gridSize, blockSize>>>(d_world, cam, image_width, image_height, d_output, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto end_time = std::chrono::high_resolution_clock::now();
    auto render_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    float total_time = render_time.count();
    float avg_time_per_pixel = total_time / static_cast<float>(total_pixels);
    float avg_time_per_row = total_time / static_cast<float>(image_height);

    std::clog << "Total render time (ms): " << total_time << "\n";
    std::clog << "Average time per row (ms): " << avg_time_per_row << "\n";
    std::clog << "Average time per pixel (ms): " << avg_time_per_pixel << "\n";

    uint8_t *h_output = new uint8_t[image_width * image_height * 3];
    checkCudaErrors(
        hipMemcpy(h_output, d_output, image_width * image_height * 3 * sizeof(uint8_t), hipMemcpyDeviceToHost));

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int i = 0; i < image_height; i++)
    { // Row
        std::clog << "\rScanlines remaining: " << (image_height - 1) << " " << std::flush;
        for (int j = 0; j < image_width; j++)
        { // Column
            int start_write_index = 3 * (i * image_width + j);
            write_color(std::cout, h_output[start_write_index], h_output[start_write_index + 1],
                        h_output[start_write_index + 2]);
        }
    }

    std::clog << "\rDone.                   \n";

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(devStates));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(cam));
    hipDeviceReset();
    delete[] h_output;
}
